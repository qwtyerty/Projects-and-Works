#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256
#define get_bin_idx(p, num_bins, s)  (int)(p.x / (double)(s/num_bins)) + (int)(p.y / (double)(s / num_bins)) * num_bins
extern double size;


/**
  GPU Parallel Implementation of particule calculations
    Simulates the interaction of n particles with a correctness check
    Each thread handles a set of particles which calculates a set of bins to calculate only
      the necessary particles who are close enough to a target particle to have any bearing on the 
      forces between each other.
**/


//
//  benchmarking program
//

class bin_t
{
  public:
    int counter; // Counter for indexing current particle
    int next_counter; // Counter for indexing particles for the next step
    int prev_counter; // Counter for indexing particles for the previous step
    int particles[]; // Indexes for particles
    int part_next[16]; // Indexes of particles for the next step
    int part_prev[16]; // Indexes of particles for the previous step

    bin_t()
    {
	  this->next_counter = 0;
	  this->prev_counter = 0;
	  this->counter = 0;
    }

    //Adds a particle id to the end
    __host__ __device__ void append(int p_id)
    {
	  this->particles[this->counter] = p_id;
	  this->counter++;
    }

    //Increments the proper counter and appends to the proper array
    __host__ __device__ void update(int new_bin, int cur_bin, int p_id)
    {
	  if(cur_bin != new_bin)
	  {
		  ++this->prev_counter;
		  this->part_prev[this->prev_counter] = p_id;
	  }
	  else
	  {
		  ++this->next_counter;
		  this->part_next[this->next_counter] = p_id;
	  }
    }

    //Resets both the next counter and the prev counter
    __host__ __device__ void reset_counters()
    {
	    this->prev_counter = this->next_counter = 0;
    }

    //Swaps next to current to start the next step
    __host__ __device__ void next(int p_id)
    {
	    this->particles[p_id] = this->part_next[p_id];
    }
};

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__global__ void compute_forces_gpu(particle_t * particles, bin_t* bins, int num_bins, int row_size ,int n)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= num_bins) return;

  int row = tid % row_size;
  int col = tid / row_size;

  for(int p = 0; p < bins[tid].counter; ++p)
  {
	  particles[bins[tid].particles[p]].ax = particles[bins[tid].particles[p]].ay = 0;
  }

  for(int r = row - 1; r <= row + 1; ++r)
  {
	  for(int c = col - 1; c <= col + 1; ++c)
	  {
		  //Bounds checking
		  if( r >= 0 && r < row_size && c >= 0 && c < row_size)
		  {

			  int nb_bin = c + r * row_size;
			  //Apply forces for each particle pair curr_part and nb_part
			  for(int curr_part = 0; curr_part < bins[tid].counter; ++curr_part)
			  {
				  for(int nb_part = 0; nb_part < bins[nb_bin].counter; ++nb_part)
				  {
					  apply_force_gpu(particles[bins[tid].particles[curr_part]],particles[bins[nb_bin].particles[nb_part]]);
				  }
			  }
		  }
	  }
  }
}



__device__ void move_gpu (particle_t  &p, double size)
{
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p.vx += p.ax * dt;
    p.vy += p.ay * dt;
    p.x  += p.vx * dt;
    p.y  += p.vy * dt;

    //
    //  bounce from walls
    //
    while( p.x < 0 || p.x > size )
    {
        p.x  = p.x < 0 ? -(p.x) : 2*size-p.x;
        p.vx = -(p.vx);
    }
    while( p.y < 0 || p.y > size )
    {
        p.y  = p.y < 0 ? -(p.y) : 2*size-p.y;
        p.vy = -(p.vy);
    }

}

__global__ void move_bins_gpu(particle_t* particles, bin_t* bins, int num_bins, int row_size, double size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_bins) return;

	bins->reset_counters();
	for(int p = 0; p < bins[tid].counter; ++p)
	{
		int next_p = bins[tid].particles[p];
		particle_t &part = particles[next_p];
		move_gpu(part, size);
		int new_bin_idx = get_bin_idx(part, row_size, size);
		bins[tid].update(new_bin_idx, tid, next_p);
	}
}

__global__ void bin_gpu(particle_t* particles, bin_t* bins, int num_bins, int row_size, double size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_bins) return;

	bins[tid].counter = bins[tid].next_counter;
	for(int p = 0; p < bins[tid].counter; ++p)
		bins[tid].next(p);
	int row = tid % row_size;
	int col = tid / row_size;
	for(int r = row - 1; r <= row + 1; ++r)
	{
		for(int c = col - 1; c <= col + 1; ++c)
		{
			if(r >= 0 && r < row_size && c >= 0 && c < row_size)
			{
				int target_bin = c + r * row_size;
				for(int p = 0; p < bins[target_bin].prev_counter; ++p)
				{
					int inc_part = bins[target_bin].part_prev[p];
					particle_t &part = particles[inc_part];
					if(get_bin_idx(part, row_size, size) == tid)
					{
						bins[tid].append(inc_part);
					}
				}
			}
		}
	}

}

int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    int row_size = size / 0.02;
    int num_bins = row_size * row_size;

    bin_t* bins = new bin_t[num_bins];
    for(int i = 0; i < n; ++i)
    {
	    int bin_idx = get_bin_idx(particles[i], row_size, size);
	    bins[bin_idx].append(i);
    } 

    bin_t* bins_gpu;
    hipMalloc((void **) &bins_gpu, num_bins * sizeof(bin_t));
    hipMemcpy(bins_gpu, bins, num_bins * sizeof(bin_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

	int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
	hipDeviceSynchronize();
	compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, bins_gpu, num_bins, row_size, n);
        //
        //  move particles
        //
	move_bins_gpu <<< blks, NUM_THREADS >>> (d_particles, bins_gpu, num_bins, row_size, size);
        //
	// Rebin for the next step
	//
	bin_gpu<<<blks, NUM_THREADS>>>(d_particles, bins_gpu, num_bins, row_size, size);

        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
	}
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    hipFree(bins_gpu);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
